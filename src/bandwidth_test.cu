/*
Pinned memory is used as a staging area for transfers from the device to the host. We can avoid 
the cost of the transfer between pageable and pinned host arrays by directly allocating our host
arrays in pinned memory. Allocate pinned host memory in CUDA C/C++ using cudaMallocHost() or 
cudaHostAlloc(), and deallocate it with cudaFreeHost(). It is possible for pinned memory allocation
to fail, so you should always check for errors. The following code excerpt demonstrates allocation 
of pinned memory with error checking.
*/


#include <hip/hip_runtime.h>
#include <iostream>
#include <assert.h>
#include <string>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline hipError_t checkCuda(hipError_t result){
#if defined(DEBUG) || defined(_DEBUG)
	if(result != cudaSuccess){
		std::cerr << "Cuda Runtime Error: " << cudaGetErrorString(result) << std::endl;
		assert(result == cudaSuccess);
	}
#endif
	return result;
}

void profileCopies(float *h_a, float *h_b, float *d, unsigned int n, std::string desc){

	std::cout << '\n' << desc <<" transfers\n";
	unsigned int bytes = n * sizeof(float);

	// Events timing
	hipEvent_t startEvent, stopEvent;

	checkCuda(hipEventCreate(&startEvent));
	checkCuda(hipEventCreate(&stopEvent));

	// Measuring transfer Host to Device
	checkCuda(hipEventRecord(startEvent, 0));
	checkCuda(hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice));
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));

	float time;
	checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
	std::cout << " Host to Device Bandwidth  (GB/s): " << (bytes * 1e-6) / time << std::endl;

	// Measuring transfer Device to Host
	checkCuda(hipEventRecord(startEvent, 0));
	checkCuda(hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost));
	checkCuda(hipEventRecord(stopEvent, 0));
	checkCuda(hipEventSynchronize(stopEvent));

	checkCuda(hipEventElapsedTime(&time, startEvent, stopEvent));
	std::cout << " Device to Host Bandwidth  (GB/s): " << (bytes * 1e-6) / time << std::endl;

	// Check result
	for(size_t i = 0; i != n; ++i){
		if(h_a[i] != h_b[i]){
			std::cout << " Transfers failed " << desc << std::endl;
			break;
		}
	}

	// Cleaning up events
	checkCuda(hipEventDestroy(startEvent));
	checkCuda(hipEventDestroy(stopEvent));
} 

int main(int argc, char* argv[]){

	uint32_t n = 4 * 1024 * 1024;
	const uint32_t bytes = n * sizeof(float);

	// Host arrays
	float *h_aPageable, *h_bPageable;
	float *h_aPinned, *h_bPinned;

	// Device array
	float *d_a;

	// Allocate and initialize
	h_aPageable = (float *)malloc(n * sizeof(float));
	h_bPageable = (float *)malloc(n * sizeof(float));

	checkCuda(hipHostMalloc((void**)&h_aPinned, bytes, hipHostMallocDefault));
	checkCuda(hipHostMalloc((void**)&h_bPinned, bytes, hipHostMallocDefault));
	
	checkCuda(hipMalloc((void**)&d_a, bytes));

	// Out device info and transfer size
	hipDeviceProp_t device;
	checkCuda(hipGetDeviceProperties(&device, 0));

	std::cout << "\n Device            : " << device.name << std::endl;
	std::cout << " Transfer size (MB): " << bytes / (1024 * 1024) << std::endl;

	// Perform copies and report results
	profileCopies(h_aPageable, h_bPageable, d_a, n, "Pageable");
	profileCopies(h_aPinned, h_bPinned, d_a, n, "Pinned");

	// Cleanup
	hipFree(d_a);
	hipHostFree(h_aPinned);
	hipHostFree(h_bPinned);
	free(h_aPageable);
	free(h_bPageable);

	return 0;
}