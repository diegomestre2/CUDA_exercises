#include "hip/hip_runtime.h"


// Device code
__global__ void array3D(hipPitchedPtr devPitchedPtr,
                         int width, int height, int depth)
{
    char* devPtr = devPitchedPtr.ptr;
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height;
    for (int z = 0; z < depth; ++z) {
        char* slice = devPtr + z * slicePitch;
        for (int y = 0; y < height; ++y) {
            float* row = (float*)(slice + y * pitch);
            for (int x = 0; x < width; ++x) {
                float element = row[x];
            }
        }
    }
}

// Device code
__global__ void array2D(float* devPtr,
                         size_t pitch, int width, int height)
{
    for (int r = 0; r < height; ++r) {
        float* row = (float*)((char*)devPtr + r * pitch);
        for (int c = 0; c < width; ++c) {
            float element = row[c];
        }
    }
}
int main(){
    // Host code
    int width = 64, height = 64, depth = 64;
    hipExtent extent = make_hipExtent(width * sizeof(float),
                                        height, depth);
    hipPitchedPtr devPitchedPtr;
    hipMalloc3D(&devPitchedPtr, extent);
    array3D<<<100, 512>>>(devPitchedPtr, width, height, depth);

    size_t pitch;
    hipMallocPitch(&devPtr, &pitch,
                width * sizeof(float), height);
    array2D<<<100, 512>>>(devPtr, pitch, width, height);
}